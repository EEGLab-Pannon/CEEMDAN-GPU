#include "hip/hip_runtime.h"
﻿/*
MIT License

Copyright (c) 2023 Electrical Brain Imaging Lab

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

/* Compile the code on linux as follows after changing the -arch parameter to mathc your target device */
//nvcc -arch=sm_70 -Xcompiler -fopenmp -lcublas -lcusparse -lcurand ./cudaCEEMDAN.cu -o CUDA_CEEMDAN

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <tuple>
#include <omp.h>
#include <dirent.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hipcub/hipcub.hpp> 
#include <hip/hip_runtime.h>
#include "statistics.h"

template <typename real_t>
__global__ void produceFirstIMF(real_t* d_IMFs, real_t* d_running, real_t* d_noisedSignal, real_t* d_currentModes, real_t* d_forNext, size_t numNoise, size_t signalLength)
{
    int samplesIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (samplesIdx < signalLength)
    {
        real_t tem = 0;
        for (int i = 0; i < numNoise; i++)
        {
            tem = tem + (1.0 / numNoise) * (d_noisedSignal[i * signalLength + samplesIdx] - d_currentModes[i * signalLength + samplesIdx]);
        }
        d_forNext[samplesIdx] = tem; // d_forNext is the medias/aux
        d_IMFs[samplesIdx] = d_running[samplesIdx] - tem;
    }
}

template <typename real_t>
__global__ void standardizeRunning(real_t* d_running, size_t SignalLength, real_t* d_singleChannelVariance)
{
    int samplesIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (samplesIdx < SignalLength)
    {
        d_running[samplesIdx] = d_running[samplesIdx] * (1.0 / sqrt(d_singleChannelVariance[0]));
    }
}

template <typename real_t>
__global__ void addNoise(real_t* d_noisedSignal, real_t* d_running, real_t* d_whiteNoiseModes, size_t SignalLength, real_t noiseStrength, real_t* d_channelVariance)
{
    int samplesIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int noiseIdx = blockIdx.y;

    if (samplesIdx < SignalLength)
    {
        d_noisedSignal[noiseIdx * SignalLength + samplesIdx] = d_running[samplesIdx] + noiseStrength * d_whiteNoiseModes[noiseIdx * SignalLength + samplesIdx] * (1.0 / sqrt(d_channelVariance[noiseIdx]));
    }
}

template <typename real_t>
__global__ void addNoise2(real_t* d_noisedSignal, real_t* d_running, real_t* d_whiteNoiseModes, size_t SignalLength, real_t noiseStrength, real_t* d_singleChannelVariance)
{
    int samplesIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int noiseIdx = blockIdx.y;

    if (samplesIdx < SignalLength)
    {
        d_noisedSignal[noiseIdx * SignalLength + samplesIdx] = d_running[samplesIdx] + noiseStrength * d_whiteNoiseModes[noiseIdx * SignalLength + samplesIdx] * sqrt(d_singleChannelVariance[0]);
    }
}

template <typename coord_t, typename real_t>
__global__ void find_extrema_shfl_max(const coord_t* d_multiChannelIndex, const real_t* d_ProjectSignals, coord_t* d_sparseMaxFlag, size_t SignalLength) {

    int channelElementsIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //int allElementsIndex = blockIdx.y * SignalLength + blockIdx.x * blockDim.x + threadIdx.x;

    int warpFlag = channelElementsIndex / 32;

    if ((channelElementsIndex - 2 * warpFlag) < SignalLength)
    {
        real_t value = d_ProjectSignals[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];
        //coord_t coord = d_multiChannelIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];

        real_t up = __shfl_up_sync(0xffffffff, value, 1);
        real_t down = __shfl_down_sync(0xffffffff, value, 1);
        if (value > up && value > down)
        {
            d_sparseMaxFlag[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = 1;
        }

        // for temporary, set edge points as it were 
        if (channelElementsIndex == 0)
        {
            d_sparseMaxFlag[blockIdx.y * SignalLength] = 1;
            d_sparseMaxFlag[blockIdx.y * SignalLength + SignalLength - 1] = 1;
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void find_extrema_shfl_min(const coord_t* d_multiChannelIndex, const real_t* d_ProjectSignals, coord_t* d_sparseMinFlag, size_t SignalLength) {

    int channelElementsIndex = blockIdx.x * blockDim.x + threadIdx.x;
    //int allElementsIndex = blockIdx.y * SignalLength + blockIdx.x * blockDim.x + threadIdx.x;

    int warpFlag = channelElementsIndex / 32;

    if ((channelElementsIndex - 2 * warpFlag) < SignalLength)
    {
        real_t value = d_ProjectSignals[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];
        //coord_t coord = d_multiChannelIndex[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)];

        real_t up = __shfl_up_sync(0xffffffff, value, 1);
        real_t down = __shfl_down_sync(0xffffffff, value, 1);

        if (value < up && value < down)
        {
            d_sparseMinFlag[blockIdx.y * SignalLength + (channelElementsIndex - 2 * warpFlag)] = 1;
        }
        // for temporary, set edge points as it were 
        if (channelElementsIndex == 0)
        {
            d_sparseMinFlag[blockIdx.y * SignalLength] = 1;
            d_sparseMinFlag[blockIdx.y * SignalLength + SignalLength - 1] = 1;
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void select_extrema_max(coord_t* d_sparseMaxFlag, real_t* d_noisedSignal, coord_t* d_noisedSignalIndex, coord_t* d_MaxScanResult, /*coord_t* d_ScanResultZeroCross,*/
    real_t* d_compactMaxValue, coord_t* d_compactMaxIndex, size_t SignalLength, coord_t* d_num_extrema_max /*coord_t* d_num_zeroCrossPoints*/)
{
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pointsIdx < SignalLength)
    {
        real_t currentValue = d_noisedSignal[noiseIdx * SignalLength + pointsIdx];
        coord_t currentIndex = d_noisedSignalIndex[noiseIdx * SignalLength + pointsIdx];
        coord_t currentFlag = d_sparseMaxFlag[noiseIdx * SignalLength + pointsIdx];
        coord_t storeLocation = d_MaxScanResult[noiseIdx * SignalLength + pointsIdx];

        if (currentFlag != 0)
        {
            d_compactMaxValue[noiseIdx * SignalLength + storeLocation] = currentValue;
            d_compactMaxIndex[noiseIdx * SignalLength + storeLocation] = currentIndex;
        }
        if (pointsIdx == SignalLength - 1)
        {
            d_num_extrema_max[noiseIdx] = storeLocation + 1;
            //d_num_zeroCrossPoints[noiseIdx] = d_ScanResultZeroCross[noiseIdx * SignalLength + pointsIdx] + 1;
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void select_extrema_min(coord_t* d_sparseMinFlag, real_t* d_noisedSignal, coord_t* d_noisedSignalIndex,
    coord_t* d_MinScanResult, real_t* d_compactMinValue, coord_t* d_compactMinIndex, size_t SignalLength,
    coord_t* d_num_extrema_min)
{
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (pointsIdx < SignalLength)
    {
        real_t currentValue = d_noisedSignal[noiseIdx * SignalLength + pointsIdx];
        coord_t currentIndex = d_noisedSignalIndex[noiseIdx * SignalLength + pointsIdx];
        coord_t currentFlag = d_sparseMinFlag[noiseIdx * SignalLength + pointsIdx];
        coord_t storeLocation = d_MinScanResult[noiseIdx * SignalLength + pointsIdx];

        if (currentFlag != 0)
        {
            d_compactMinValue[noiseIdx * SignalLength + storeLocation] = currentValue;
            d_compactMinIndex[noiseIdx * SignalLength + storeLocation] = currentIndex;
        }

        if (pointsIdx == SignalLength - 1)
        {
            d_num_extrema_min[noiseIdx] = storeLocation + 1;
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void setBoundaryMax(real_t* d_compactMaxValue, coord_t* d_compactMaxIndex, coord_t* d_MaxScanResult, size_t SignalLength)
{
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.x;
    int pointsIdx = threadIdx.x;

    if ((pointsIdx == 0) && (d_MaxScanResult[noiseIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_max, t_max;

        coord_t storeLocation_max = d_MaxScanResult[noiseIdx * SignalLength + pointsIdx];
        coord_t loc_max = noiseIdx * SignalLength + storeLocation_max;

        slope_max = (d_compactMaxValue[loc_max + 2] - d_compactMaxValue[loc_max + 1]) / (d_compactMaxIndex[loc_max + 2] - d_compactMaxIndex[loc_max + 1]);
        t_max = d_compactMaxValue[loc_max + 1] - slope_max * (d_compactMaxIndex[loc_max + 1] - d_compactMaxIndex[loc_max]);

        if (t_max > d_compactMaxValue[loc_max])
        {
            d_compactMaxValue[loc_max] = t_max;
        }
    }

    if ((pointsIdx == 1) && (d_MaxScanResult[noiseIdx * SignalLength + SignalLength - 1] > 4))
    {

        real_t slope_max, t_max;

        coord_t storeLocation_max = d_MaxScanResult[noiseIdx * SignalLength + SignalLength - 1];
        coord_t loc_max = noiseIdx * SignalLength + storeLocation_max;

        slope_max = (d_compactMaxValue[loc_max - 1] - d_compactMaxValue[loc_max - 2]) / (d_compactMaxIndex[loc_max - 1] - d_compactMaxIndex[loc_max - 2]);
        t_max = d_compactMaxValue[loc_max - 1] + slope_max * (d_compactMaxIndex[loc_max] - d_compactMaxIndex[loc_max - 1]);

        if (t_max > d_compactMaxValue[loc_max])
        {
            d_compactMaxValue[loc_max] = t_max;
        }
    }
}

template <typename coord_t, typename real_t>
__global__ void setBoundaryMin(real_t* d_compactMinValue, coord_t* d_compactMinIndex, coord_t* d_MinScanResult, size_t SignalLength)
{
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.x;
    int pointsIdx = threadIdx.x;

    if ((pointsIdx == 0) && (d_MinScanResult[noiseIdx * SignalLength + SignalLength - 1] > 4))
    {
        real_t slope_min, t_min;

        coord_t storeLocation_min = d_MinScanResult[noiseIdx * SignalLength + pointsIdx];
        coord_t loc_min = noiseIdx * SignalLength + storeLocation_min;

        slope_min = (d_compactMinValue[loc_min + 2] - d_compactMinValue[loc_min + 1]) / (d_compactMinIndex[loc_min + 2] - d_compactMinIndex[loc_min + 1]);
        t_min = d_compactMinValue[loc_min + 1] - slope_min * (d_compactMinIndex[loc_min + 1] - d_compactMinIndex[loc_min]);

        if (t_min < d_compactMinValue[loc_min])
        {
            d_compactMinValue[loc_min] = t_min;
        }
    }

    if ((pointsIdx == 1) && (d_MinScanResult[noiseIdx * SignalLength + SignalLength - 1] > 4))
    {

        real_t slope_min, t_min;

        coord_t storeLocation_min = d_MinScanResult[noiseIdx * SignalLength + SignalLength - 1];
        coord_t loc_min = noiseIdx * SignalLength + storeLocation_min;

        slope_min = (d_compactMinValue[loc_min - 1] - d_compactMinValue[loc_min - 2]) / (d_compactMinIndex[loc_min - 1] - d_compactMinIndex[loc_min - 2]);
        t_min = d_compactMinValue[loc_min - 1] + slope_min * (d_compactMinIndex[loc_min] - d_compactMinIndex[loc_min - 1]);

        if (t_min < d_compactMinValue[loc_min])
        {
            d_compactMinValue[loc_min] = t_min;
        }
    }
}

template <typename real_t>
__global__ void preSetTridiagonalMatrix(real_t* d_upperDia, real_t* d_middleDia, real_t* d_lowerDia, real_t* d_right, size_t signalLnegth)
{
    int noiseIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = noiseIdx * signalLnegth + pointsIdx;

    if (pointsIdx < signalLnegth)
    {
        d_upperDia[idx] = 0;
        d_lowerDia[idx] = 0;
        d_right[idx] = 0;
        d_middleDia[idx] = 1;
    }

}

// for natural boundary conditions
template <typename coord_t, typename real_t>
__global__ void tridiagonal_setup(coord_t* d_num_extrema, coord_t* d_extrema_x, real_t* d_extrema_y, real_t* d_upper_dia, real_t* d_middle_dia, real_t* d_lower_dia, real_t* d_right_dia, size_t SignalLength) {
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    //int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = noiseIdx * SignalLength + pointsIdx;
    const int num_equation = d_num_extrema[noiseIdx];
    int idxForRight = noiseIdx * SignalLength + pointsIdx; // to compact the y value within one direction vector
    if (pointsIdx == 0)
    {
        d_middle_dia[idx] = 1;
        d_upper_dia[idx] = 0;
        d_lower_dia[idx] = 0;
        d_right_dia[idxForRight] = 0; // it used to be idx
    }
    if (pointsIdx == num_equation - 1)
    {
        d_middle_dia[idx] = 1;
        d_lower_dia[idx] = 0;
        d_upper_dia[idx] = 0;
        d_right_dia[idxForRight] = 0;
    }
    if (pointsIdx != 0 && pointsIdx < num_equation - 1)
    {
        d_middle_dia[idx] = 2 * (((d_extrema_x[idx] - d_extrema_x[idx - 1]) + (d_extrema_x[idx + 1] - d_extrema_x[idx])));
        d_upper_dia[idx] = d_extrema_x[idx + 1] - d_extrema_x[idx];
        d_lower_dia[idx] = d_extrema_x[idx] - d_extrema_x[idx - 1];
        d_right_dia[idxForRight] = 3 * ((d_extrema_y[idx + 1] - d_extrema_y[idx]) / (d_extrema_x[idx + 1] - d_extrema_x[idx]) -
            (d_extrema_y[idx] - d_extrema_y[idx - 1]) / (d_extrema_x[idx] - d_extrema_x[idx - 1]));
    }
}

// for not-a-knot boundary conditions
template <typename coord_t, typename real_t>
__global__ void tridiagonal_setup_nak(coord_t* d_num_extrema, coord_t* d_extrema_x, real_t* d_extrema_y, real_t* d_upper_dia, real_t* d_middle_dia, real_t* d_lower_dia, real_t* d_right_dia, size_t SignalLength, size_t SignalDim, size_t NumDirVector) {
    int dirVecIdx = blockIdx.z;
    int signalDimIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    //int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = dirVecIdx * SignalDim * SignalLength + signalDimIdx * SignalLength + pointsIdx;
    const int num_equation = d_num_extrema[dirVecIdx] - 1;
    if (pointsIdx == 0)
    {
        real_t h0 = d_extrema_x[idx + 1] - d_extrema_x[idx];
        real_t h1 = d_extrema_x[idx + 2] - d_extrema_x[idx + 1];
        d_middle_dia[idx] = -h1 * h1 + h0 * h0;
        d_upper_dia[idx] = (h0 + h1) * h1 + 2 * h0 * (h0 + h1);
        d_lower_dia[idx] = 0; // fixed
        d_right_dia[idx] = 3 * (h0 / h1 * (d_extrema_y[idx + 2] - d_extrema_y[idx + 1]) - d_extrema_y[idx + 1] + d_extrema_y[idx]);
    }

    if (pointsIdx == num_equation)
    {
        real_t hn_2 = d_extrema_x[idx - 1] - d_extrema_x[idx - 2];
        real_t hn_1 = d_extrema_x[idx] - d_extrema_x[idx - 1];
        d_middle_dia[idx] = -hn_2 * hn_2 + hn_1 * hn_1;
        d_lower_dia[idx] = (hn_2 + hn_1) * hn_2 + hn_1 * 2 * (hn_2 + hn_1);
        d_upper_dia[idx] = 0; // fixed
        d_right_dia[idx] = 3 * ((d_extrema_y[idx] - d_extrema_y[idx - 1]) - hn_1 / hn_2 * (d_extrema_y[idx - 1] - d_extrema_y[idx - 2]));
    }
    if (pointsIdx != 0 && pointsIdx < num_equation)
    {
        d_middle_dia[idx] = 2 * (((d_extrema_x[idx] - d_extrema_x[idx - 1]) + (d_extrema_x[idx + 1] - d_extrema_x[idx])));
        d_upper_dia[idx] = d_extrema_x[idx + 1] - d_extrema_x[idx];
        d_lower_dia[idx] = d_extrema_x[idx] - d_extrema_x[idx - 1];
        d_right_dia[idx] = 3 * (d_extrema_y[idx + 1] - d_extrema_y[idx]) / (d_extrema_x[idx + 1] - d_extrema_x[idx]) - 3 * (d_extrema_y[idx] - d_extrema_y[idx - 1]) / (d_extrema_x[idx] - d_extrema_x[idx - 1]);
    }
}

template <typename coord_t, typename real_t>
__global__ void spline_coefficients(const real_t* a, real_t* b, real_t* c, real_t* d, coord_t* extrema_points_x, size_t SignalLength, coord_t* d_num_extrema, real_t* solution) {
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = noiseIdx * SignalLength + pointsIdx;

    const int num_equation = d_num_extrema[noiseIdx];
    int idxForSolution = noiseIdx * SignalLength + pointsIdx; // to index the compacted sulution matrix, it used to be idx

    if (pointsIdx < num_equation - 1)
    {
        real_t h = extrema_points_x[idx + 1] - extrema_points_x[idx];
        b[idx] = (a[idx + 1] - a[idx]) / h - h * (2 * solution[idxForSolution] + solution[idxForSolution + 1]) / 3;
        d[idx] = (solution[idxForSolution + 1] - solution[idxForSolution]) / (3 * h);
        c[idx] = solution[idxForSolution];
    }

}

template <typename coord_t, typename real_t>
__global__ void interpolate(const real_t* a, real_t* b, real_t* c, real_t* d, coord_t* d_envelopeIndex, real_t* d_envelopeValue, coord_t* d_extremaIndex, size_t SignalLength, coord_t* d_num_extrema) {
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = noiseIdx * SignalLength + pointsIdx;
    int idx2 = noiseIdx * SignalLength;

    int num_coefs = d_num_extrema[noiseIdx];
    size_t num_samples = SignalLength;
    if (pointsIdx < num_samples)// what the hell here? <= or <
    {
        //int i = 0;
        int coef_idx = 0;
        int low = 1;
        int high = num_coefs - 1;
        // binary search for coef index
        while (low <= high) {
            int mid = (low + high) / 2;
            if ((pointsIdx > d_extremaIndex[idx2 + mid - 1]) && (pointsIdx <= d_extremaIndex[idx2 + mid])) {
                coef_idx = mid - 1;
                break;
            }
            else if (pointsIdx < d_extremaIndex[idx2 + mid]) {
                high = mid - 1;
            }
            else {
                low = mid + 1;
            }
        }

        coord_t t = d_envelopeIndex[idx] - d_extremaIndex[idx2 + coef_idx];
        d_envelopeValue[idx] = a[idx2 + coef_idx] + (b[idx2 + coef_idx] + (c[idx2 + coef_idx] + d[idx2 + coef_idx] * t) * t) * t;
    }

}

template <typename coord_t, typename real_t>
__global__ void averageUppperLower(real_t* d_meanEnvelope, real_t* d_upperEnvelope, real_t* d_lowerEnvelope, size_t SignalLength, coord_t* d_num_extrema_max, coord_t* d_num_extrema_min)
{
    //int dirVecIdx = blockIdx.z;
    //int signalDimIdx = blockIdx.y;
    int noiseIdx = blockIdx.y;
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = noiseIdx * SignalLength + pointsIdx;

    if ((d_num_extrema_max[noiseIdx] > 3) && (d_num_extrema_min[noiseIdx] > 3) && (pointsIdx < SignalLength))
    {
        d_meanEnvelope[idx] = (d_upperEnvelope[idx] + d_lowerEnvelope[idx]) / 2.0;
    }
}

template <typename real_t>
__global__ void produceSX(real_t* d_sxVector, real_t* d_upperEnvelope, real_t* d_lowerEnvelope, size_t SignalLength)
{
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int noiseIdx = blockIdx.y;
    int idx = noiseIdx * SignalLength + pointsIdx;

    if (pointsIdx < SignalLength)
    {
        d_sxVector[idx] = abs((d_upperEnvelope[idx] + d_lowerEnvelope[idx]) / (d_upperEnvelope[idx] - d_lowerEnvelope[idx]));
    }
}

template <typename real_t>
__global__ void thresholdJudge(real_t* d_sxVector, real_t* d_channelMark, real_t threshold_1, real_t threshold_2, size_t signalLength)
{
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int noiseIdx = blockIdx.y;

    if (pointsIdx < signalLength)
    {
        if (d_sxVector[noiseIdx * signalLength + pointsIdx] > threshold_2)
        {
            d_channelMark[noiseIdx] = 1;
        }

        if (d_sxVector[noiseIdx * signalLength + pointsIdx] > threshold_1)
        {
            d_sxVector[noiseIdx * signalLength + pointsIdx] = 1;
        }
        else
        {
            d_sxVector[noiseIdx * signalLength + pointsIdx] = 0;
        }
    }

}

template <typename coord_t, typename real_t>
__global__ void siftingCriterion(real_t* d_finishFlag, real_t* d_realizationMark, real_t* d_channelMeans, real_t* d_channelMark, real_t threshold_3, coord_t* d_num_extrema_max, coord_t* d_num_extrema_min, size_t numNoise, size_t idxIter, size_t maxIter)
{
    int noiseIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (noiseIdx < numNoise)
    {
        int criterion1 = d_channelMeans[noiseIdx] > threshold_3 || d_channelMark[noiseIdx] && (d_num_extrema_max[noiseIdx] + d_num_extrema_min[noiseIdx]) > 6; // 2 + 4
        //int criterion2 = abs(d_num_zeroCrossPoints[noiseIdx] - d_num_extrema_max[noiseIdx] - d_num_extrema_min[noiseIdx] + 4) > 2; // 1 + 4
        //if (d1 || (d1 ^ d2))
        if (criterion1)
        {
            d_realizationMark[noiseIdx] = 1;
            if (d_finishFlag[noiseIdx] == 0 && idxIter == (maxIter - 1))
            {
                d_realizationMark[noiseIdx] = 0;
                d_finishFlag[noiseIdx] = 1;
            }
        }
        else
        {
            if (d_finishFlag[noiseIdx] == 0)
            {
                d_realizationMark[noiseIdx] = 0;
                d_finishFlag[noiseIdx] = 1;
            }
            else
            {
                d_realizationMark[noiseIdx] = 1;
            }
        }
    }
}

template <typename real_t>
__global__ void updateRealizations(real_t* d_realizationMark, real_t* currentWhiteNoiseModes, real_t* d_noisedSignal, real_t* d_meanEnvelope, size_t SignalLength, size_t numNoise, size_t j, size_t max_iter)
{
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int noiseIdx = blockIdx.y;

    if (pointsIdx < SignalLength)
    {
        if ((d_realizationMark[noiseIdx] == 1))
        {
            d_noisedSignal[noiseIdx * SignalLength + pointsIdx] = d_noisedSignal[noiseIdx * SignalLength + pointsIdx] - d_meanEnvelope[noiseIdx * SignalLength + pointsIdx];
        }
        if ((d_realizationMark[noiseIdx] == 0))
        {
            currentWhiteNoiseModes[noiseIdx * SignalLength + pointsIdx] = d_noisedSignal[noiseIdx * SignalLength + pointsIdx];
        }
    }
}

template <typename real_t>
__global__ void checkBreak(real_t* d_finishFlag, int* whetherStopSifting, size_t numNoise)
{
    extern __shared__ int s_finishFlag[];
    int noiseIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int index = threadIdx.x;

    if (noiseIdx < blockDim.x)
    {
        s_finishFlag[index] = 1;
    }
    __syncthreads();

    if (noiseIdx < numNoise)
    {
        s_finishFlag[index] = d_finishFlag[noiseIdx] == 1;
        __syncthreads();

        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (index < s)
            {
                s_finishFlag[index] &= s_finishFlag[index + s];
            }
            __syncthreads();
        }

        if (index == 0)
        {
            //whetherStopSifting[0] &= s_finishFlag[0];
            whetherStopSifting[0] = s_finishFlag[0];
        }
    }

}

template <typename real_t>
__global__ void produceResidue(real_t* d_noisedSignal, real_t* d_currentModes, real_t* d_residue, size_t SignalLength)
{
    int pointsIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int noiseIdx = blockIdx.y;
    if (pointsIdx < SignalLength)
    {
        d_residue[noiseIdx * SignalLength + pointsIdx] = d_noisedSignal[noiseIdx * SignalLength + pointsIdx] - d_currentModes[noiseIdx * SignalLength + pointsIdx];
    }
}

template <typename real_t>
__global__ void averageUpdateSignal(real_t* d_residue, real_t* d_forNext, real_t* d_IMFs, size_t numNoise, size_t SignalLength, size_t imfIdx)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (SignalLength))
    {
        real_t tem = 0;
        for (int i = 0; i < numNoise; i++)
        {
            tem = tem + d_residue[i * SignalLength + idx];
        }
        d_IMFs[imfIdx * SignalLength + idx] = d_forNext[idx] - tem / real_t(numNoise);
        d_forNext[idx] = tem / real_t(numNoise);
    }
}

template <typename real_t>
__global__ void updateSignal(real_t* d_current, real_t* d_whiteNoise, size_t numNoise, size_t SignalLength)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < (numNoise * SignalLength))
    {
        d_whiteNoise[idx] = d_whiteNoise[idx] - d_current[idx];
    }
}

int getBinSize(char* path)
{
    int  size = 0;
    FILE* fp = fopen(path, "rb");
    if (fp)
    {
        fseek(fp, 0, SEEK_END);
        size = ftell(fp);
        fclose(fp);
    }
    //printf("\npath=%s,size=%d \n", path, size);
    return size;
}

void readBin(char* path, char* buf, size_t size)
{
    FILE* infile;
    if ((infile = fopen(path, "rb")) == NULL)
    {
        printf("\nCan not open the path: %s \n", path);
        exit(-1);
    }
    fread(buf, sizeof(char), size, infile);
    fclose(infile);
}

void writeBin(char* path, char* buf, size_t size)
{
    FILE* outfile;
    if ((outfile = fopen(path, "wb")) == NULL)
    {
        printf("\nCan not open the path: %s \n", path);
        exit(-1);
    }
    fwrite(buf, sizeof(char), size, outfile);
    fclose(outfile);
}

template <typename coord_t, typename real_t>
double ceemdan(size_t numNoise, size_t SignalLength, size_t num_IMFs, size_t max_iter, coord_t* d_x, real_t* d_y, real_t* d_IMFs, real_t noiseStrength)
{
    //===============load data===============
    real_t* d_noisedSignal = NULL;
    hipMalloc((void**)&d_noisedSignal, numNoise * SignalLength * sizeof(real_t));
    real_t* d_running = NULL;
    hipMalloc((void**)&d_running, SignalLength * sizeof(real_t));

    //===============preparation for noise===============
    real_t* d_whiteNoise = NULL;
    hipMalloc((void**)&d_whiteNoise, numNoise * SignalLength * sizeof(real_t));
    hiprandGenerator_t gen;
    real_t meanValue = 0.0;
    real_t stddev = 1.0;

    real_t* d_whiteNoiseModes = NULL;
    hipMalloc((void**)&d_whiteNoiseModes, num_IMFs * numNoise * SignalLength * sizeof(real_t));
    real_t* d_current = NULL;
    hipMalloc((void**)&d_current, numNoise * SignalLength * sizeof(real_t));

    real_t* d_currentModes = NULL;
    hipMalloc((void**)&d_currentModes, numNoise * SignalLength * sizeof(real_t));

    real_t* d_channelMeans = NULL;
    hipMalloc((void**)&d_channelMeans, numNoise * sizeof(real_t));
    real_t* d_channelVariance = NULL;
    hipMalloc((void**)&d_channelVariance, numNoise * sizeof(real_t));

    real_t* d_singleChannelMean = NULL;
    hipMalloc((void**)&d_singleChannelMean, sizeof(real_t));
    real_t* d_singleChannelVariance = NULL;
    hipMalloc((void**)&d_singleChannelVariance, sizeof(real_t));

    //===============preparation for extreme points detection===============
    coord_t* d_sparseFlag;
    hipMalloc((void**)&d_sparseFlag, numNoise * SignalLength * sizeof(coord_t));
    coord_t* d_sparseZeroCrossFlag;
    hipMalloc((void**)&d_sparseZeroCrossFlag, numNoise * SignalLength * sizeof(coord_t));

    coord_t* d_noisedSignalIndex = NULL;
    hipMalloc((void**)&d_noisedSignalIndex, numNoise * SignalLength * sizeof(coord_t));
    for (int i = 0; i < numNoise; i++)
    {
        int head = i * SignalLength;
        hipMemcpy((d_noisedSignalIndex + head), d_x, SignalLength * sizeof(coord_t), hipMemcpyDeviceToDevice);
    }

    //===============preparation for prefix sum===============
    coord_t* d_ScanResult;
    hipMalloc((void**)&d_ScanResult, numNoise * SignalLength * sizeof(coord_t));

    void* d_temStorage = NULL;
    size_t tempStorageBytes = 128 * 1024 * 1024 * sizeof(real_t);
    hipMalloc(&d_temStorage, tempStorageBytes);

    //===============preparation for extreme points select===============
    real_t* d_compactValue;
    coord_t* d_compactIndex;
    coord_t* d_num_extrema_max, * d_num_extrema_min; // *d_num_zeroCrossPoints;

    hipMalloc((void**)&d_compactValue, numNoise * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_compactIndex, numNoise * SignalLength * sizeof(coord_t));
    hipMalloc((void**)&d_num_extrema_max, numNoise * sizeof(coord_t));
    hipMalloc((void**)&d_num_extrema_min, numNoise * sizeof(coord_t));

    //===============preparation for tridiagonal setting===============
    real_t* d_upperDia = NULL, * d_middleDia = NULL, * d_lowerDia = NULL, * d_right = NULL;

    hipMalloc((void**)&d_upperDia, numNoise * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_middleDia, numNoise * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_lowerDia, numNoise * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_right, numNoise * SignalLength * sizeof(real_t));

    //===============preparation for tridiagonal solving===============

    real_t* currentUpperDia = NULL;
    real_t* currentMiddleDia = NULL;
    real_t* currentLowerDia = NULL;
    real_t* currentRightDia = NULL;
    real_t* currentSolution = NULL;
    coord_t* h_num_extrema = (coord_t*)malloc(numNoise * sizeof(coord_t));

    hipsparseHandle_t handle_sparse;
    hipsparseCreate(&handle_sparse);
    size_t* buffer_size = (size_t*)malloc(sizeof(size_t));
    real_t* buffer = NULL;
    hipMalloc(&buffer, 128 * 1024 * 1024 * sizeof(real_t));

    //===============preparation for interpolate values===============
    real_t* d_envelopeVauleMax = NULL, * d_envelopeVauleMin = NULL;
    hipMalloc((void**)&d_envelopeVauleMax, numNoise * SignalLength * sizeof(real_t));
    hipMalloc((void**)&d_envelopeVauleMin, numNoise * SignalLength * sizeof(real_t));

    //===============preparation for averaging upper and lower===============
    real_t* d_meanEnvelope = NULL;
    hipMalloc((void**)&d_meanEnvelope, numNoise * SignalLength * sizeof(real_t));

    real_t* d_forNext = NULL;
    hipMalloc((void**)&d_forNext, SignalLength * sizeof(real_t));

    real_t* d_residue = NULL;
    hipMalloc((void**)&d_residue, numNoise * SignalLength * sizeof(real_t));

    //===============preparation for CUDA graphs===============
    hipStream_t streamForGraphPreFix; // maybe we can capture on default stream?
    hipStream_t streamForGraphTriSolve;

    hipStreamCreate(&streamForGraphPreFix);
    hipStreamCreate(&streamForGraphTriSolve);

    hipGraph_t graphPreFix;
    hipGraphExec_t graph_execPreFix;
    int isCapturedPrfFix = 0;

    hipGraph_t graphTriSolve;
    hipGraphExec_t graph_execTriSolve;
    hipsparseSetStream(handle_sparse, streamForGraphTriSolve);
    int isCapturedTriSolve = 0;

    //===============generate white noise===============
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    hiprandGenerateNormal(gen, d_whiteNoise, numNoise * SignalLength, meanValue, stddev);

    //===============preparation for sifting stop===============
    real_t threshold_1 = 0.05; // sd
    real_t threshold_2 = 0.5; // sd2
    real_t threshold_3 = 0.05; // tol

    real_t* d_sxVector = NULL;
    hipMalloc((void**)&d_sxVector, numNoise * SignalLength * sizeof(real_t));

    real_t* d_realizationMark = NULL;
    hipMalloc((void**)&d_realizationMark, numNoise * sizeof(real_t));
    hipMemset(d_realizationMark, 0, numNoise * sizeof(real_t));

    real_t* d_chanelMeansSiftingStop = NULL;
    hipMalloc((void**)&d_chanelMeansSiftingStop, numNoise * sizeof(real_t));

    real_t* d_channelMark = NULL;
    hipMalloc((void**)&d_channelMark, numNoise * sizeof(real_t));

    real_t* d_finishFlag = NULL;
    hipMalloc((void**)&d_finishFlag, numNoise * sizeof(real_t));

    int* whetherStopSifting = NULL;
    hipHostMalloc((void**)&whetherStopSifting, sizeof(int), hipHostMallocMapped);

    //===============preparation for grid and block dim===============
    dim3 blockDimShfl(256);
    size_t numThreads = (SignalLength / 30 + 1) * 32;
    dim3 gridDimShfl(numThreads / blockDimShfl.x + (numThreads % blockDimShfl.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimSelectExtrema(256);
    dim3 gridDimSelectExtrema(SignalLength / blockDimSelectExtrema.x + (SignalLength % blockDimSelectExtrema.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimSetBoundary(2);
    dim3 gridDimSetBoundary(numNoise);

    dim3 blockDimPreSetTri(256);
    dim3 gridDimPreSetTri(SignalLength / blockDimPreSetTri.x + (SignalLength % blockDimPreSetTri.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimTriSet(256);
    dim3 gridDimTriSet(SignalLength / blockDimTriSet.x + (SignalLength % blockDimTriSet.x == 0 ? 0 : 1), numNoise); // too much idle threads

    dim3 blockDimSplineCoe(256);
    dim3 gridDimSplineCoe(SignalLength / blockDimSplineCoe.x + (SignalLength % blockDimSplineCoe.x == 0 ? 0 : 1), numNoise); // too much idle threads

    dim3 blockDimInterpolate(256);
    dim3 gridDimInterpolate(SignalLength / blockDimInterpolate.x + (SignalLength % blockDimInterpolate.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimMeanEnvelope(256);
    dim3 gridDimMeanEnvelope(SignalLength / blockDimInterpolate.x + (SignalLength % blockDimInterpolate.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimProduceSX(256);
    dim3 gridDimProduceSX(SignalLength / blockDimInterpolate.x + (SignalLength % blockDimInterpolate.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimThresholdJudge(256);
    dim3 gridDimThresholdJudge(SignalLength / blockDimInterpolate.x + (SignalLength % blockDimInterpolate.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimMultiply(256);
    dim3 gridDimMultiply(numNoise / blockDimMultiply.x + (numNoise % blockDimMultiply.x == 0 ? 0 : 1));

    dim3 blockDimSiftingCriterion(256);
    dim3 gridDimSiftingCriterion(numNoise / blockDimSiftingCriterion.x + (numNoise % blockDimSiftingCriterion.x == 0 ? 0 : 1));

    dim3 blockDimUpdateRealizations(256);
    dim3 gridDimUpdateRealizations(SignalLength / blockDimUpdateRealizations.x + (SignalLength % blockDimUpdateRealizations.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimCheckBreak(256);
    dim3 gridDimCheckBreak(numNoise / blockDimCheckBreak.x + (numNoise % blockDimCheckBreak.x == 0 ? 0 : 1));

    dim3 blockDimAddNoiseSingleChannel(256);
    dim3 gridDimAddNoiseSingleChannel(SignalLength / blockDimAddNoiseSingleChannel.x + +(SignalLength % blockDimAddNoiseSingleChannel.x == 0 ? 0 : 1), 1);

    dim3 blockDimStandardize(256);
    dim3 gridDimStandardize(SignalLength / blockDimStandardize.x + +(SignalLength % blockDimStandardize.x == 0 ? 0 : 1), 1);

    dim3 blockDimAddNoise(256);
    dim3 gridDimAddNoise(SignalLength / blockDimAddNoise.x + +(SignalLength % blockDimAddNoise.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimProduceResidue(256);
    dim3 gridDimProduceResidue(SignalLength / blockDimProduceResidue.x + (SignalLength % blockDimProduceResidue.x == 0 ? 0 : 1), numNoise);

    dim3 blockDimAverageUpdateSignal(256);
    dim3 gridDimAverageUpdateSignal(SignalLength / blockDimAverageUpdateSignal.x + (SignalLength % blockDimAverageUpdateSignal.x == 0 ? 0 : 1));

    dim3 blockDimProduceFirstIMF(256);
    dim3 gridDimProduceFirstIMF(SignalLength / blockDimProduceFirstIMF.x + (SignalLength % blockDimProduceFirstIMF.x == 0 ? 0 : 1));

    dim3 blockDimUpdateSignal(256);
    dim3 gridDimUpdateSignal(numNoise * SignalLength / blockDimUpdateSignal.x + (numNoise * SignalLength % blockDimUpdateSignal.x == 0 ? 0 : 1));

    //=============================================
    //         generate white noise modes
    //=============================================

    //==========replace the d_whiteNoise with a fixed one==========
    //char filePathInput[] = "E:\\Software\\Nvidia_Development\\Project\\CUDA_CEEMDAN\\CEEMDAN\\ceemd_paper_example\\white_noise.bin";
    //real_t* h_whiteNoise = (real_t*)malloc(numNoise * SignalLength * sizeof(real_t));
    //char* buf = (char*)malloc(numNoise * SignalLength * sizeof(real_t));
    //readBin(filePathInput, buf, numNoise* SignalLength * sizeof(real_t));
    //h_whiteNoise = (float*)buf;
    //hipMemcpy(d_whiteNoise, h_whiteNoise, numNoise* SignalLength * sizeof(real_t), hipMemcpyHostToDevice);
    //==========replace the d_whiteNoise with a fixed one==========

    for (size_t i = 0; i < num_IMFs - 1; ++i)
    {
        hipMemcpy(d_current, d_whiteNoise, numNoise * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);
        hipMemset(d_finishFlag, 0, numNoise * sizeof(real_t));
        *whetherStopSifting = 1;

        for (size_t j = 0; j < 2000; ++j) // default is 2000
        {
            //==================extreme points detection max============
            hipMemset(d_sparseFlag, 0, numNoise * SignalLength * sizeof(coord_t));
            hipMemset(d_channelMark, 0, numNoise * sizeof(real_t));
            find_extrema_shfl_max << <gridDimShfl, blockDimShfl >> > (d_noisedSignalIndex, d_current, d_sparseFlag, SignalLength);

            //==================prefix scan max======================
            if (isCapturedPrfFix == 0)
            {
                hipStreamBeginCapture(streamForGraphPreFix, hipStreamCaptureModeGlobal);
                for (size_t k = 0; k < numNoise; k++)
                {
                    int offset = k * SignalLength;
                    hipcub::DeviceScan::ExclusiveSum(d_temStorage, tempStorageBytes, d_sparseFlag + offset, d_ScanResult + offset, SignalLength, streamForGraphPreFix);
                }
                hipStreamEndCapture(streamForGraphPreFix, &graphPreFix);
                hipDeviceSynchronize();
                hipGraphInstantiateWithFlags(&graph_execPreFix, graphPreFix, 0); // it was 0
                hipGraphLaunch(graph_execPreFix, streamForGraphPreFix);
                isCapturedPrfFix = 1;
            }
            else
            {
                hipGraphLaunch(graph_execPreFix, streamForGraphPreFix);
                hipStreamSynchronize(streamForGraphPreFix);
            }

            for (size_t k = 0; k < numNoise; k++)
            {
                int offset = k * SignalLength;
                hipcub::DeviceScan::ExclusiveSum(d_temStorage, tempStorageBytes, d_sparseFlag + offset, d_ScanResult + offset, SignalLength, streamForGraphPreFix);
            }

            //==================extreme points select max============
            select_extrema_max << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_noisedSignalIndex,
                d_ScanResult, /*d_ScanResultZeroCross,*/ d_compactValue, d_compactIndex, SignalLength, d_num_extrema_max /*d_num_zeroCrossPoints*/);

            setBoundaryMax << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                d_ScanResult, SignalLength);

            //==================set up tridiagonal matrix max============
            preSetTridiagonalMatrix << <gridDimPreSetTri, blockDimPreSetTri >> > (d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

            tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_max, d_compactIndex, d_compactValue,
                d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

            //==================solve tridiagonal matrix max============
            if (isCapturedTriSolve == 0)
            {
                hipStreamBeginCapture(streamForGraphTriSolve, hipStreamCaptureModeGlobal);
                for (size_t k = 0; k < numNoise; k++)
                {
                    currentUpperDia = d_upperDia + k * SignalLength;
                    currentMiddleDia = d_middleDia + k * SignalLength;
                    currentLowerDia = d_lowerDia + k * SignalLength;
                    currentSolution = d_right + k * SignalLength;
					
                    hipsparseSgtsv2_nopivot(handle_sparse, SignalLength * 0.5 + 2/*pow(0.9, i + 1)*/, 1, currentLowerDia, currentMiddleDia, currentUpperDia, currentSolution, SignalLength * 0.5 + 2/*pow(0.9, i + 1)*/, buffer);
                }
                hipStreamEndCapture(streamForGraphTriSolve, &graphTriSolve);
                hipDeviceSynchronize();
                hipGraphInstantiateWithFlags(&graph_execTriSolve, graphTriSolve, 0); // it was 0
                hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                isCapturedTriSolve = 1;
            }
            else
            {
                hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                hipStreamSynchronize(streamForGraphTriSolve);
            }

            //==================compute spline coefficients max============
            spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalLength, d_num_extrema_max, d_right);

            //==================interpolate values max============
            interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_noisedSignalIndex, d_envelopeVauleMax, d_compactIndex, SignalLength, d_num_extrema_max);

            //==================extreme points detection min============
            hipMemset(d_sparseFlag, 0, numNoise * SignalLength * sizeof(coord_t));
            find_extrema_shfl_min << <gridDimShfl, blockDimShfl >> > (d_noisedSignalIndex, d_current, d_sparseFlag, SignalLength);

            //==================prefix scan min============
            hipGraphLaunch(graph_execPreFix, streamForGraphPreFix);
            hipStreamSynchronize(streamForGraphPreFix);

            //==================extreme points select min============
            select_extrema_min << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_noisedSignalIndex,
                d_ScanResult, d_compactValue, d_compactIndex, SignalLength, d_num_extrema_min);

            setBoundaryMin << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                d_ScanResult, SignalLength);

            //==================set up tridiagonal matrix min============
            preSetTridiagonalMatrix << <gridDimPreSetTri, blockDimPreSetTri >> > (d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

            tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_min, d_compactIndex, d_compactValue,
                d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

            //==================solve tridiagonal matrix min============
            hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
            hipStreamSynchronize(streamForGraphTriSolve);

            //==================compute spline coefficients min============
            spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalLength, d_num_extrema_min, d_right);

            //==================interpolate values min============
            interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_noisedSignalIndex, d_envelopeVauleMin, d_compactIndex, SignalLength, d_num_extrema_min);

            //==================average upper and lower============
            averageUppperLower << <gridDimMeanEnvelope, blockDimMeanEnvelope >> > (d_meanEnvelope, d_envelopeVauleMax, d_envelopeVauleMin, SignalLength, d_num_extrema_max, d_num_extrema_min);

            //==================sifting stop critierion============
            // amp = abs(envmax-envmin)/2;
            // envmoy = (envmin+envmax)/2;
            // sx = abs(envmoy)./amp;
            // so, sx = abs(envmin+envmax) / abs(envmin-envmax)
            produceSX << <gridDimProduceSX, blockDimProduceSX >> > (d_sxVector, d_envelopeVauleMax, d_envelopeVauleMin, SignalLength);

            //sx > sd; produce a boolean matrix (samples * channels) and store back in d_sxVector 
            //any(sx > sd2); produce a boolean vector (1 * channels) and store in d_channelMark
            thresholdJudge << <gridDimThresholdJudge, blockDimThresholdJudge >> > (d_sxVector, d_channelMark, threshold_1, threshold_2, SignalLength);

            //mean(sx > sd); calculate the mean of the bollean matrix for each channel and store in d_chanelMeansSiftingStop
            mean << <gridDimMeanEnvelope, blockDimMeanEnvelope, numNoise * sizeof(real_t) >> > (d_sxVector, SignalLength, numNoise, d_chanelMeansSiftingStop);
            multiply << <gridDimMultiply, blockDimMultiply >> > (d_chanelMeansSiftingStop, d_chanelMeansSiftingStop, (1.0 / SignalLength), numNoise);

            // (mean(sx > sd) > tol | any(sx > sd2)) & (all(nem > 2)); store the result in d_realizationMark
            siftingCriterion << <gridDimSiftingCriterion, blockDimSiftingCriterion >> > (d_finishFlag, d_realizationMark, d_chanelMeansSiftingStop, d_channelMark, threshold_3, d_num_extrema_max, d_num_extrema_min, numNoise, j, 2000); // deafault max iteration number is 2000

            //==================update each realization or export d_current as IMF============
            updateRealizations << <gridDimUpdateRealizations, blockDimUpdateRealizations >> > (d_realizationMark, (d_whiteNoiseModes + i * numNoise * SignalLength), d_current, d_meanEnvelope, SignalLength, numNoise, j, max_iter);

            //==================check break============
            checkBreak << <gridDimCheckBreak, blockDimCheckBreak, 256 * sizeof(int) >> > (d_finishFlag, whetherStopSifting, numNoise);
            hipDeviceSynchronize();

            if (whetherStopSifting[0] == 1)
				break;
        }

        updateSignal << <gridDimUpdateSignal, blockDimUpdateSignal >> > ((d_whiteNoiseModes + i * numNoise * SignalLength), d_whiteNoise, numNoise, SignalLength);
    }
    hipMemcpy(&d_whiteNoiseModes[(num_IMFs - 1) * numNoise * SignalLength], d_whiteNoise, numNoise * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);

    //=============================================
    //                 CEEMDAN
    //=============================================

    //===============IMF loop===============
    hipMemcpy(d_running, d_y, SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);
    double start_time = omp_get_wtime();
    for (size_t i = 0; i < num_IMFs - 1; ++i)
    {
        hipMemset(d_finishFlag, 0, numNoise * sizeof(real_t));
        *whetherStopSifting = 1;

        if (i == 0)
        {
            //===============noise adding and signal standardization===============
            mean << <gridDimAddNoiseSingleChannel, blockDimAddNoiseSingleChannel, 1 * sizeof(real_t) >> > (d_running, SignalLength, 1, d_singleChannelMean);
            multiply << <1, 1 >> > (d_singleChannelMean, d_singleChannelMean, (1.0 / SignalLength), 1);

            variance << <gridDimAddNoiseSingleChannel, blockDimAddNoiseSingleChannel, 1 * sizeof(real_t) >> > (d_running, SignalLength, 1, d_singleChannelMean, d_singleChannelVariance);
            multiply << <1, 1 >> > (d_singleChannelVariance, d_singleChannelVariance, (1.0 / (SignalLength - 1)), 1);

            mean << <gridDimAddNoise, blockDimAddNoise, numNoise * sizeof(real_t) >> > ((d_whiteNoiseModes + i * numNoise * SignalLength), SignalLength, numNoise, d_channelMeans);
            multiply << <1, numNoise >> > (d_channelMeans, d_channelMeans, (1.0 / SignalLength), numNoise);

            variance << <gridDimAddNoise, blockDimAddNoise, numNoise * sizeof(real_t) >> > ((d_whiteNoiseModes + i * numNoise * SignalLength), SignalLength, numNoise, d_channelMeans, d_channelVariance);
            multiply << <1, numNoise >> > (d_channelVariance, d_channelVariance, (1.0 / (SignalLength - 1)), numNoise);

            standardizeRunning << <gridDimStandardize, blockDimStandardize >> > (d_running, SignalLength, d_singleChannelVariance);
            addNoise << <gridDimAddNoise, blockDimAddNoise >> > (d_noisedSignal, d_running, d_whiteNoiseModes + i * numNoise * SignalLength, SignalLength, noiseStrength, d_channelVariance);

            //===============sifting loop===============
            hipMemcpy(d_current, d_noisedSignal, numNoise * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);

            for (size_t j = 0; j < max_iter; ++j)
            {
                hipMemset(d_channelMark, 0, numNoise * sizeof(real_t));

                //==================extreme points detection max============
                hipMemset(d_sparseFlag, 0, numNoise * SignalLength * sizeof(coord_t));
                find_extrema_shfl_max << <gridDimShfl, blockDimShfl >> > (d_noisedSignalIndex, d_current, d_sparseFlag, SignalLength);

                //==================prefix scan max======================
                hipGraphLaunch(graph_execPreFix, streamForGraphPreFix);
                hipStreamSynchronize(streamForGraphPreFix);

                //==================extreme points select max============
                select_extrema_max << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_noisedSignalIndex,
                    d_ScanResult, d_compactValue, d_compactIndex, SignalLength, d_num_extrema_max);

                setBoundaryMax << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                    d_ScanResult, SignalLength);

                //==================set up tridiagonal matrix max============
                preSetTridiagonalMatrix << <gridDimPreSetTri, blockDimPreSetTri >> > (d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_max, d_compactIndex, d_compactValue,
                    d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                //==================solve tridiagonal matrix max============
                if (isCapturedTriSolve == 0)
                {
                    hipStreamBeginCapture(streamForGraphTriSolve, hipStreamCaptureModeGlobal);
                    for (size_t k = 0; k < numNoise; k++)
                    {
                        currentUpperDia = d_upperDia + k * SignalLength;
                        currentMiddleDia = d_middleDia + k * SignalLength;
                        currentLowerDia = d_lowerDia + k * SignalLength;
                        currentSolution = d_right + k * SignalLength;
                        hipsparseSgtsv2_nopivot(handle_sparse, SignalLength * pow(0.5, i + 1), 1, currentLowerDia, currentMiddleDia, currentUpperDia, currentSolution, SignalLength * pow(0.5, i + 1), buffer);
                    }
                    hipStreamEndCapture(streamForGraphTriSolve, &graphTriSolve);
                    hipDeviceSynchronize();
                    hipGraphInstantiateWithFlags(&graph_execTriSolve, graphTriSolve, 0); // it was 0
                    hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                    isCapturedTriSolve = 1;
                }
                else
                {
                    hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                    hipStreamSynchronize(streamForGraphTriSolve);
                }

                //==================compute spline coefficients max============
                spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalLength, d_num_extrema_max, d_right);

                //==================interpolate values max============
                interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_noisedSignalIndex, d_envelopeVauleMax, d_compactIndex, SignalLength, d_num_extrema_max);

                //==================extreme points detection min============
                hipMemset(d_sparseFlag, 0, numNoise * SignalLength * sizeof(coord_t));
                find_extrema_shfl_min << <gridDimShfl, blockDimShfl >> > (d_noisedSignalIndex, d_current, d_sparseFlag, SignalLength);

                //==================prefix scan min============
                hipGraphLaunch(graph_execPreFix, streamForGraphPreFix);
                hipStreamSynchronize(streamForGraphPreFix);

                //==================extreme points select min============
                select_extrema_min << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_noisedSignalIndex,
                    d_ScanResult, d_compactValue, d_compactIndex, SignalLength, d_num_extrema_min);

                setBoundaryMin << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                    d_ScanResult, SignalLength);

                //==================set up tridiagonal matrix min============
                preSetTridiagonalMatrix << <gridDimPreSetTri, blockDimPreSetTri >> > (d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_min, d_compactIndex, d_compactValue,
                    d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                //==================solve tridiagonal matrix min============
                hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                hipStreamSynchronize(streamForGraphTriSolve);

                //==================compute spline coefficients min============
                spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalLength, d_num_extrema_min, d_right);

                //==================interpolate values min============
                interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_noisedSignalIndex, d_envelopeVauleMin, d_compactIndex, SignalLength, d_num_extrema_min);

                //==================average upper and lower============
                averageUppperLower << <gridDimMeanEnvelope, blockDimMeanEnvelope >> > (d_meanEnvelope, d_envelopeVauleMax, d_envelopeVauleMin, SignalLength, d_num_extrema_max, d_num_extrema_min);

                //==================sifting stop critierion============
                // amp = abs(envmax-envmin)/2;
                // envmoy = (envmin+envmax)/2;
                // sx = abs(envmoy)./amp;
                // so, sx = abs(envmin+envmax) / abs(envmin-envmax)
                produceSX << <gridDimProduceSX, blockDimProduceSX >> > (d_sxVector, d_envelopeVauleMax, d_envelopeVauleMin, SignalLength);

                //sx > sd; produce a boolean matrix (samples * channels) and store back in d_sxVector 
                //any(sx > sd2); produce a boolean vector (1 * channels) and store in d_channelMark
                thresholdJudge << <gridDimThresholdJudge, blockDimThresholdJudge >> > (d_sxVector, d_channelMark, threshold_1, threshold_2, SignalLength);

                //mean(sx > sd); calculate the mean of the bollean matrix for each channel and store in d_chanelMeansSiftingStop
                mean << <gridDimMeanEnvelope, blockDimMeanEnvelope, numNoise * sizeof(real_t) >> > (d_sxVector, SignalLength, numNoise, d_chanelMeansSiftingStop);
                multiply << <gridDimMultiply, blockDimMultiply >> > (d_chanelMeansSiftingStop, d_chanelMeansSiftingStop, (1.0 / SignalLength), numNoise);

                // (mean(sx > sd) > tol | any(sx > sd2)) & (all(nem > 2)); store the result in d_realizationMark
                siftingCriterion << <gridDimSiftingCriterion, blockDimSiftingCriterion >> > (d_finishFlag, d_realizationMark, d_chanelMeansSiftingStop, d_channelMark, threshold_3, d_num_extrema_max, d_num_extrema_min, numNoise, j, max_iter);

                //==================update each realization or export d_current as IMF============
                updateRealizations << <gridDimUpdateRealizations, blockDimUpdateRealizations >> > (d_realizationMark, d_currentModes, d_current, d_meanEnvelope, SignalLength, numNoise, j, max_iter);

                //==================check break====================
                checkBreak << <gridDimCheckBreak, blockDimCheckBreak, 256 * sizeof(int) >> > (d_finishFlag, whetherStopSifting, numNoise);
                hipDeviceSynchronize();

                if (whetherStopSifting[0] == 1)
                {
                    break;
                }
            }
            produceFirstIMF << <gridDimProduceFirstIMF, blockDimProduceFirstIMF >> > (d_IMFs, d_running, d_noisedSignal, d_currentModes, d_forNext, numNoise, SignalLength);
        }
        else
        {
            //===============noise adding===============
            mean << <gridDimAddNoiseSingleChannel, blockDimAddNoiseSingleChannel, 1 * sizeof(real_t) >> > (d_forNext, SignalLength, 1, d_singleChannelMean);
            multiply << <1, 1 >> > (d_singleChannelMean, d_singleChannelMean, (1.0 / SignalLength), 1);

            variance << <gridDimAddNoiseSingleChannel, blockDimAddNoiseSingleChannel, 1 * sizeof(real_t) >> > (d_forNext, SignalLength, 1, d_singleChannelMean, d_singleChannelVariance);
            multiply << <1, 1 >> > (d_singleChannelVariance, d_singleChannelVariance, (1.0 / (SignalLength - 1)), 1);

            addNoise2 << <gridDimAddNoise, blockDimAddNoise >> > (d_noisedSignal, d_forNext, d_whiteNoiseModes + i * numNoise * SignalLength, SignalLength, noiseStrength, d_singleChannelVariance);

            //===============sifting loop===============
            hipMemcpy(d_current, d_noisedSignal, numNoise * SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);
            for (size_t j = 0; j < max_iter; ++j)
            {
                hipMemset(d_channelMark, 0, numNoise * sizeof(real_t));

                //==================extreme points detection max============
                hipMemset(d_sparseFlag, 0, numNoise * SignalLength * sizeof(coord_t));
                find_extrema_shfl_max << <gridDimShfl, blockDimShfl >> > (d_noisedSignalIndex, d_current, d_sparseFlag, SignalLength);

                //==================prefix scan max======================
                hipGraphLaunch(graph_execPreFix, streamForGraphPreFix);
                hipStreamSynchronize(streamForGraphPreFix);

                //==================extreme points select max============
                select_extrema_max << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_noisedSignalIndex,
                    d_ScanResult, d_compactValue, d_compactIndex, SignalLength, d_num_extrema_max);

                setBoundaryMax << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                    d_ScanResult, SignalLength);

                //==================set up tridiagonal matrix max============
                preSetTridiagonalMatrix << <gridDimPreSetTri, blockDimPreSetTri >> > (d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_max, d_compactIndex, d_compactValue,
                    d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                //==================solve tridiagonal matrix max============
                if (isCapturedTriSolve == 0)
                {
                    hipStreamBeginCapture(streamForGraphTriSolve, hipStreamCaptureModeGlobal);
                    for (size_t k = 0; k < numNoise; k++)
                    {
                        currentUpperDia = d_upperDia + k * SignalLength;
                        currentMiddleDia = d_middleDia + k * SignalLength;
                        currentLowerDia = d_lowerDia + k * SignalLength;
                        currentSolution = d_right + k * SignalLength;
						
                        hipsparseSgtsv2_nopivot(handle_sparse, SignalLength * pow(0.9, i + 1), 1, currentLowerDia, currentMiddleDia, currentUpperDia, currentSolution, SignalLength * pow(0.9, i + 1), buffer);
                    }
                    hipStreamEndCapture(streamForGraphTriSolve, &graphTriSolve);
                    hipDeviceSynchronize();
                    hipGraphInstantiateWithFlags(&graph_execTriSolve, graphTriSolve, 0); // it was 0
                    hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                    isCapturedTriSolve = 1;
                }
                else
                {
                    hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                    hipStreamSynchronize(streamForGraphTriSolve);
                }

                //==================compute spline coefficients max============
                spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalLength, d_num_extrema_max, d_right);

                //==================interpolate values max============
                interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_noisedSignalIndex, d_envelopeVauleMax, d_compactIndex, SignalLength, d_num_extrema_max);

                //==================extreme points detection min============
                hipMemset(d_sparseFlag, 0, numNoise * SignalLength * sizeof(coord_t));
                find_extrema_shfl_min << <gridDimShfl, blockDimShfl >> > (d_noisedSignalIndex, d_current, d_sparseFlag, SignalLength);

                //==================prefix scan min============
                hipGraphLaunch(graph_execPreFix, streamForGraphPreFix);
                hipStreamSynchronize(streamForGraphPreFix);

                //==================extreme points select min============
                select_extrema_min << <gridDimSelectExtrema, blockDimSelectExtrema >> > (d_sparseFlag, d_current, d_noisedSignalIndex,
                    d_ScanResult, d_compactValue, d_compactIndex, SignalLength, d_num_extrema_min);

                setBoundaryMin << <gridDimSetBoundary, blockDimSetBoundary >> > (d_compactValue, d_compactIndex,
                    d_ScanResult, SignalLength);

                //==================set up tridiagonal matrix min============
                preSetTridiagonalMatrix << <gridDimPreSetTri, blockDimPreSetTri >> > (d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                tridiagonal_setup << <gridDimTriSet, blockDimTriSet >> > (d_num_extrema_min, d_compactIndex, d_compactValue,
                    d_upperDia, d_middleDia, d_lowerDia, d_right, SignalLength);

                //==================solve tridiagonal matrix min============     
                hipGraphLaunch(graph_execTriSolve, streamForGraphTriSolve);
                hipStreamSynchronize(streamForGraphTriSolve);

                //==================compute spline coefficients min============
                spline_coefficients << <gridDimSplineCoe, blockDimSplineCoe >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_compactIndex, SignalLength, d_num_extrema_min, d_right);

                //==================interpolate values min============
                interpolate << <gridDimInterpolate, blockDimInterpolate >> > (d_compactValue, d_upperDia, d_middleDia, d_lowerDia, d_noisedSignalIndex, d_envelopeVauleMin, d_compactIndex, SignalLength, d_num_extrema_min);

                //==================average upper and lower============
                averageUppperLower << <gridDimMeanEnvelope, blockDimMeanEnvelope >> > (d_meanEnvelope, d_envelopeVauleMax, d_envelopeVauleMin, SignalLength, d_num_extrema_max, d_num_extrema_min);

                //==================sifting stop critierion============
                 // amp = abs(envmax-envmin)/2;
                 // envmoy = (envmin+envmax)/2;
                 // sx = abs(envmoy)./amp;
                 // so, sx = abs(envmin+envmax) / abs(envmin-envmax)
                produceSX << <gridDimProduceSX, blockDimProduceSX >> > (d_sxVector, d_envelopeVauleMax, d_envelopeVauleMin, SignalLength);

                //sx > sd; produce a boolean matrix (samples * channels) and store back in d_sxVector 
                //any(sx > sd2); produce a boolean vector (1 * channels) and store in d_channelMark
                thresholdJudge << <gridDimThresholdJudge, blockDimThresholdJudge >> > (d_sxVector, d_channelMark, threshold_1, threshold_2, SignalLength);

                //mean(sx > sd); calculate the mean of the bollean matrix for each channel and store in d_chanelMeansSiftingStop
                mean << <gridDimMeanEnvelope, blockDimMeanEnvelope, numNoise * sizeof(real_t) >> > (d_sxVector, SignalLength, numNoise, d_chanelMeansSiftingStop);
                multiply << <gridDimMultiply, blockDimMultiply >> > (d_chanelMeansSiftingStop, d_chanelMeansSiftingStop, (1.0 / SignalLength), numNoise);

                // (mean(sx > sd) > tol | any(sx > sd2)) & (all(nem > 2)); store the result in d_realizationMark
                siftingCriterion << <gridDimSiftingCriterion, blockDimSiftingCriterion >> > (d_finishFlag, d_realizationMark, d_chanelMeansSiftingStop, d_channelMark, threshold_3, d_num_extrema_max, d_num_extrema_min, numNoise, j, max_iter);

                //==================update each realization or export d_current as IMF============
                updateRealizations << <gridDimUpdateRealizations, blockDimUpdateRealizations >> > (d_realizationMark, d_currentModes, d_current, d_meanEnvelope, SignalLength, numNoise, j, max_iter);

                //==================check break====================
                checkBreak << <gridDimCheckBreak, blockDimCheckBreak, 256 * sizeof(int) >> > (d_finishFlag, whetherStopSifting, numNoise);
                hipDeviceSynchronize();

                if (whetherStopSifting[0] == 1)
                {
                    break;
                }
            }

            produceResidue << <gridDimProduceResidue, blockDimProduceResidue >> > (d_noisedSignal, d_currentModes, d_residue, SignalLength);
            averageUpdateSignal << <gridDimAverageUpdateSignal, blockDimAverageUpdateSignal >> > (d_residue, d_forNext, d_IMFs, numNoise, SignalLength, i);

            //=============================TEST============================
            //real_t* h_currentModes = (real_t*)malloc(SignalLength * sizeof(real_t));
            //hipMemcpy(h_currentModes, d_currentModes, SignalLength * sizeof(real_t), hipMemcpyDeviceToHost);
            //char test_file_h_currentModes[] = "./h_currentModes.bin";
            //writeBin(test_file_h_currentModes, (char*)h_currentModes, SignalLength * sizeof(real_t));
            //=============================TEST============================
        }
    }

    double end_time = omp_get_wtime();
    double diff = end_time - start_time;
    hipMemcpy(&d_IMFs[(num_IMFs - 1) * SignalLength], d_forNext, SignalLength * sizeof(real_t), hipMemcpyDeviceToDevice);

    //free all the CPU and GPU memory here
    hipFree(d_noisedSignal);
    hipFree(d_running);
    hipFree(d_sparseFlag);
    hipFree(d_ScanResult);
    hipFree(d_compactValue);
    hipFree(d_compactIndex);
    hipFree(d_num_extrema_max);
    hipFree(d_num_extrema_min);
    hipFree(d_upperDia);
    hipFree(d_middleDia);
    hipFree(d_lowerDia);
    hipFree(d_right);
    hipFree(d_envelopeVauleMax);
    hipFree(d_envelopeVauleMin);
    hipFree(d_meanEnvelope);
    free(h_num_extrema);
    hipFree(buffer);
    hipFree(d_whiteNoise);
    hipFree(d_whiteNoiseModes);
    hipFree(d_current);
    hipFree(d_currentModes);
    hipFree(d_channelMeans);
    hipFree(d_channelVariance);
    hipFree(d_singleChannelMean);
    hipFree(d_singleChannelVariance);
    hipFree(d_sparseZeroCrossFlag);
    hipFree(d_noisedSignalIndex);
    hipFree(d_forNext);
    hipFree(d_residue);
    hipFree(d_sxVector);
    hipFree(d_realizationMark);
    hipFree(d_chanelMeansSiftingStop);
    hipFree(d_channelMark);
    hipFree(d_finishFlag);
    hipFree(d_temStorage);

    return diff;
}


void runBenchmark()
{
    //int maxIterationsIdx = 50;
    //int numRealizationsIdx = 50;
    //int sigLenIdx = 118;
    float noiseStrength = 0.2;
    size_t num_IMFs = 13;
    char filePath[] = "./eegSampleDataCH1_244032.bin";
    int maxIterations[55] = {10, 10, 20, 50, 100, 100, 200, 300, 400, 500, 600, 700, 800, 900, 1000,
                              1100,	1200, 1300,	1400, 1500,	1600, 1700,	1800, 1900,	2000,
                              2100, 2200, 2300, 2400, 2500, 2600, 2700, 2800, 2900, 3000,
                              3100, 3200, 3300, 3400, 3500, 3600, 3700, 3800, 3900, 4000,
                              4100, 4200, 4300, 4400, 4500, 4600, 4700, 4800, 4900, 5000};

    int numRealizaitons[51] = { 2, 10,	20,	30,	40,	50,	60,	70,	80,	90,	100,
                                110, 120, 130, 140, 150, 160, 170, 180, 190, 200,
                                210, 220, 230, 240, 250, 260, 270, 280, 290, 300,
                                310, 320, 330, 340, 350, 360, 370, 380, 390, 400,
                                410, 420, 430, 440, 450, 460, 470, 480, 490, 500};

    int signalLength[119] = { 2049, 2049, 4097, 6145, 8193, 10241, 12289, 14337, 16385, 18433, 20481,
                            22529, 24577, 26625, 28673, 30721, 32769, 34817, 36865, 38913, 40961,
                            43009, 45057, 47105, 49153,	51201, 53249, 55297, 57345,	59393, 61441,
                            63489, 65537, 67585, 69633,	71681, 73729, 75777, 77825,	79873, 81921,
                            83969, 86017, 88065, 90113,	92161, 94209, 96257, 98305,	100353, 102401,
                            104449,	106497,	108545, 110593, 112641, 114689,	116737,	118785,	120833,
                            122881,	124929,	126977,	129025,	131073,	133121,	135169,	137217,	139265,
                            141313,	143361, 145409, 147457, 149505,	151553,	153601,	155649,	157697,
                            159745,	161793,	163841,	165889,	167937,	169985,	172033,	174081, 176129,
                            178177,	180225,	182273,	184321,	186369,	188417,	190465,	192513,	194561,
                            196609,	198657,	200705,	202753,	204801,	206849,	208897,	210945,	212993,
                            215041,	217089,	219137, 221185,	223233,	225281,	227329,	229377,	231425,
                            233473,	235521,	237569,	239617,	241665};

    for (int sigLenIdx = 50; sigLenIdx <= 50; sigLenIdx = sigLenIdx + 5)
    {
        float* h_y, * d_y;
        int* h_x, * d_x;

        // get data size
        const size_t SignalLength = signalLength[sigLenIdx];
        size_t oneChannelNbytes_coord = SignalLength * sizeof(int);
        size_t oneChannelNbytes_real = SignalLength * sizeof(float);

        // allocate array on host
        h_x = (int*)malloc(oneChannelNbytes_coord);
        h_y = (float*)malloc(oneChannelNbytes_real);

        // allocate array on device
        hipMalloc((void**)&d_x, oneChannelNbytes_coord);
        hipMalloc((void**)&d_y, oneChannelNbytes_real);

        // load data
        char* buf = (char*)malloc(oneChannelNbytes_real);
        readBin(filePath, buf, oneChannelNbytes_real);
        h_y = (float*)buf;

        // generate data index
        for (size_t i = 0; i < SignalLength; i++) {
            h_x[i] = (int)i;
        }

        // copy data to device
        hipMemcpy(d_x, h_x, oneChannelNbytes_coord, hipMemcpyHostToDevice);
        hipMemcpy(d_y, h_y, oneChannelNbytes_real, hipMemcpyHostToDevice);

        // allocate memory for output
        float* d_IMFs = NULL;
        hipMalloc(&d_IMFs, num_IMFs * SignalLength * sizeof(float));
        float* IMFs = (float*)malloc(num_IMFs * SignalLength * sizeof(float));

        //ceemdan processing
        for (int maxIterationsIdx = 1; maxIterationsIdx <= 3; maxIterationsIdx = maxIterationsIdx + 1)
        {
            for (int numRealizationsIdx = 50; numRealizationsIdx <= 50; numRealizationsIdx = numRealizationsIdx + 5)
            {
                FILE* fp = fopen("printout.txt", "a+");
                size_t numNoise = numRealizaitons[numRealizationsIdx];
                size_t max_iter = maxIterations[maxIterationsIdx];
                double exeTime = ceemdan(numNoise, SignalLength, num_IMFs, max_iter, d_x, d_y, d_IMFs, noiseStrength);
                printf("numMaxIterations: %d; numNoise: %d; signalLength: %d; Execution time: %f \n", max_iter, numNoise, SignalLength, exeTime);
                fprintf(fp, "numMaxIterations: %d; numNoise: %d; signalLength: %d; Execution time: %f \n", max_iter, numNoise, SignalLength, exeTime);
                fclose(fp);
            }
        }
        hipFree(d_y);
        hipFree(d_x);
        hipFree(d_IMFs);
    }
}

int main()
{
//    runBenchmark();

    //============================= Single run of CEEMDAN =============================

    //configuration for the input signal
    size_t numNoise = 20;
    size_t num_IMFs = 3;
    size_t max_iter = 300;
    float noiseStrength = 0.2;

    float* h_y, * d_y;
    int* h_x, * d_x;

    // get data size
    char filePathInput[] = "./eegSampleDataCH4.bin";
    size_t nbytes = getBinSize(filePathInput); // in bytes
    const size_t SignalLength = nbytes / sizeof(float);
    size_t oneChannelNbytes_coord = SignalLength * sizeof(int);
    size_t oneChannelNbytes_real = SignalLength * sizeof(float);

    // allocate array on host
    h_x = (int*)malloc(oneChannelNbytes_coord);
    h_y = (float*)malloc(oneChannelNbytes_real);

    // allocate array on device
    hipMalloc((void**)&d_x, oneChannelNbytes_coord);
    hipMalloc((void**)&d_y, oneChannelNbytes_real);

    // load data
    char* buf = (char*)malloc(oneChannelNbytes_real);
    readBin(filePathInput, buf, oneChannelNbytes_real);
    h_y = (float*)buf;

    // generate data index
    for (size_t i = 0; i < SignalLength; i++) {
        h_x[i] = (int)i;
    }

    // copy data to device
    hipMemcpy(d_x, h_x, oneChannelNbytes_coord, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, oneChannelNbytes_real, hipMemcpyHostToDevice);

    //ceemdan processing
    float* d_IMFs = NULL;
    hipMalloc(&d_IMFs, num_IMFs * SignalLength * sizeof(float));
    float* IMFs = (float*)malloc(num_IMFs * SignalLength * sizeof(float));

    double exeTime = ceemdan(numNoise, SignalLength, num_IMFs, max_iter, d_x, d_y, d_IMFs, noiseStrength);

    printf("Execution time: %f \n", exeTime);
    hipMemcpy(IMFs, d_IMFs, num_IMFs * SignalLength * sizeof(float), hipMemcpyDeviceToHost);
    char IMFs_file[] = "./modes.bin";
    writeBin(IMFs_file, (char*)IMFs, num_IMFs * SignalLength * sizeof(float));
}